#include "hip/hip_runtime.h"
#include "CUDAProcess.h"

#include "hip/hip_runtime.h"
#include ""

#include <cmath>

__device__ float cuda_osc_next(float d_phaseIncArray, float twoPI, float mPI, bool muted, OscillatorMode mode){

	float value = 0.0;
	if (muted){
		return value;
	}
	switch (mode)
	{
	case OSCILLATOR_MODE_SINE:
		value = sin(d_phaseIncArray);
		break;
	case OSCILLATOR_MODE_SAW:
		value = 1.0 - (2.0 * d_phaseIncArray / twoPI);
		break;
	case OSCILLATOR_MODE_SQUARE:
		if (d_phaseIncArray <= mPI) {
			value = 1.0;
		}
		else {
			value = -1.0;
		}
		break;
	case OSCILLATOR_MODE_TRIANGLE:
		value = -1.0 + (2.0 * d_phaseIncArray / twoPI);
		value = 2.0 * (fabs(value) - 0.5);
		break;
	default:
		break;
	}

	return value;

}

__global__ void cuda_process(float* d_lbuffer, float* d_rbuffer, float* d_phaseIncArray, int d_velocity, float d_lastfreq, float twoPI, float mPI, 
	OscillatorMode mode){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//float d_osc_freq = 0.0;
	bool d_osc_muted = false;

	if (d_velocity > 0){
		//d_osc_freq = d_lastfreq;
		d_osc_muted = false;
	}
	else {
		d_osc_muted = true;
	}

	d_lbuffer[idx] = d_rbuffer[idx] = cuda_osc_next(d_phaseIncArray[idx], twoPI, mPI, d_osc_muted, mode) * d_velocity / 127.0;

}

void CUDAProcess::Process(double* lbuffer, double* rbuffer, int lastvel, double lastfreq, double mPI, double twoPI, 
	double mSampleRate, Envelope* mEnvelope, OscillatorMode mode){

	const int cuda_nFrames = 256;

	// convert double buffer to float fbuffer
	// TODO REMOVE this with compute capability graphics card >= 2.0
	float h_lbuffer[cuda_nFrames];
	float h_rbuffer[cuda_nFrames];
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		h_lbuffer[i] = (float)lbuffer[i];
		h_rbuffer[i] = (float)rbuffer[i];
	}

	// calculate al phaseIncrements
	double phaseIncArray[cuda_nFrames];
	double mPhaseIncrement_aux = lastfreq * 2 * mPI / mSampleRate;
	// get previous array last value
	double mPhase_aux = lastPhaseIncrementValue + mPhaseIncrement_aux;
	//double mPhase_aux = mOscillator.getmPhase();
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		phaseIncArray[i] = mPhase_aux;
		mPhase_aux += mPhaseIncrement_aux;
		while (mPhase_aux >= twoPI) {
			mPhase_aux -= twoPI;
		}
	}

	// TODO REMOVE this with compute capability graphics card >= 2.0
	float phaseIncArray_f[cuda_nFrames];
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		phaseIncArray_f[i] = (float)phaseIncArray[i];
	}

	// ---------------------------------------- CUDA ------------------------------------------------
	float* d_lbuffer;
	float* d_rbuffer;
	float* d_phaseIncArray;
	
	int d_size = cuda_nFrames*sizeof(float);
	hipMalloc((void **)&d_lbuffer, d_size);
	hipMalloc((void **)&d_rbuffer, d_size);
	hipMalloc((void **)&d_phaseIncArray, d_size);

	hipMemcpy(d_lbuffer, h_lbuffer, d_size, hipMemcpyHostToDevice);
	hipMemcpy(d_rbuffer, h_rbuffer, d_size, hipMemcpyHostToDevice);
	hipMemcpy(d_phaseIncArray, phaseIncArray_f, d_size, hipMemcpyHostToDevice);

	// change depending on available CUDA cores
	int block_size = 4;
	int n_blocks = cuda_nFrames / block_size + (cuda_nFrames % block_size == 0 ? 0 : 1);

	cuda_process <<< n_blocks, block_size >>>(d_lbuffer, d_rbuffer, d_phaseIncArray, lastvel, (float)lastfreq, (float)twoPI, (float)mPI, mode);

	hipMemcpy(h_lbuffer, d_lbuffer, d_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_rbuffer, d_rbuffer, d_size, hipMemcpyDeviceToHost);

	hipFree(d_lbuffer);
	hipFree(d_rbuffer);
	hipFree(d_phaseIncArray);

	// copy float fbuffer to double buffer
	// TODO REMOVE this with compute capability graphics card >= 2.0
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		lbuffer[i] = (double)h_lbuffer[i];
		rbuffer[i] = (double)h_rbuffer[i];
	}

	// calculate envelop stages
	double envelopeStage[cuda_nFrames];
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		envelopeStage[i] = mEnvelope->nextSample();
	}

	// add envelope volume to buffers
	for (size_t i = 0; i < cuda_nFrames; i++)
	{
		lbuffer[i] *= envelopeStage[i];
		rbuffer[i] *= envelopeStage[i];
	}
	
	
	/*
	for (int i = 0; i < cuda_nFrames; ++i) {

		if (lastvel > 0) {
			mOscillator.setFrequency(lastfreq);
			mOscillator.setMuted(false);
		}
		else {
			mOscillator.setMuted(true);
		}

		//lbuffer[i] = rbuffer[i] = mFilter.process(mOscillator.nextSample() * mEnvelope->nextSample() * lastvel / 127.0);
		//lbuffer[i] = rbuffer[i] = mOscillator.nextSample(phaseIncArray_f[i]) * lastvel / 127.0;
		lbuffer[i] = rbuffer[i] = mOscillator.nextSample(phaseIncArray[i]) * lastvel / 127.0;
	}
	*/
	
	// save last increment value for next chunk
	lastPhaseIncrementValue = phaseIncArray[cuda_nFrames - 1];

}

