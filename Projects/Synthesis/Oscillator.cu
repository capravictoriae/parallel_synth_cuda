#include "hip/hip_runtime.h"
#include "Oscillator.h"

#include "MIDIReceiver.h"

#include <vector>

#include "hip/hip_runtime.h"
#include ""

__global__ void generate_oscillator_sine(float *d_buffer, float* d_phaseIncArray, OscillatorMode mode, float mPI, float twoPI){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	switch (mode)
	{
	case OSCILLATOR_MODE_SINE:
		d_buffer[idx] = sin(d_phaseIncArray[idx]);
		break;
	case OSCILLATOR_MODE_SAW:
		d_buffer[idx] = 1.0 - (2.0 * d_phaseIncArray[idx] / twoPI);
		break;
	case OSCILLATOR_MODE_SQUARE:
		if (d_phaseIncArray[idx] <= mPI) {
			d_buffer[idx] = 1.0;
		}
		else {
			d_buffer[idx] = -1.0;
		}
		break;
	case OSCILLATOR_MODE_TRIANGLE:
		double value = -1.0 + (2.0 * d_phaseIncArray[idx] / twoPI);
		d_buffer[idx] = 2.0 * (fabs(value) - 0.5);
		break;
	default:
		break;
	}
}

void Oscillator::setMode(OscillatorMode mode) {
    mOscillatorMode = mode;
}

void Oscillator::setFrequency(double frequency) {	
    mFrequency = frequency;
    updateIncrement();
}

void Oscillator::setSampleRate(double sampleRate) {
    mSampleRate = sampleRate;
    updateIncrement();
}

void Oscillator::updateIncrement() {
    mPhaseIncrement = mFrequency * 2 * mPI / mSampleRate;
}

void Oscillator::generate(double* buffer, int nFrames) {
    const double twoPI = 2 * mPI;
	const int nFrames_size = 256;
	
	// convert double buffer to float fbuffer
	float fbuffer[nFrames_size];
	for (size_t i = 0; i < nFrames_size; i++)
	{
		fbuffer[i] = (float)buffer[i];
	}

	// calculate al phaseIncrements
	float phaseIncArray[nFrames_size];
	for (size_t i = 0; i < nFrames_size; i++)
	{
		phaseIncArray[i] = mPhase;
		mPhase += mPhaseIncrement;
		while (mPhase >= twoPI) {
			mPhase -= twoPI;
		}
	}

	
	// ----------------------- CUDA ----------------------------

	float* d_buffer;
	float* d_phaseIncArray;

	int d_size = nFrames*sizeof(float);
	hipMalloc((void **) &d_buffer, d_size);
	hipMalloc((void **) &d_phaseIncArray, d_size);

	hipMemcpy(d_buffer, fbuffer, d_size, hipMemcpyHostToDevice);
	hipMemcpy(d_phaseIncArray, phaseIncArray, d_size, hipMemcpyHostToDevice);

	int block_size = 4;
	int n_blocks = nFrames / block_size + (nFrames % block_size == 0 ? 0 : 1);

	generate_oscillator_sine << < n_blocks, block_size >> >(d_buffer, d_phaseIncArray, mOscillatorMode, (float)mPI, (float)twoPI);

	hipMemcpy(fbuffer, d_buffer, d_size, hipMemcpyDeviceToHost);

	hipFree(d_buffer);

	// copy float fbuffer to double buffer
	for (size_t i = 0; i < nFrames_size; i++)
	{
		buffer[i] = (double)fbuffer[i];
	}
	

	// ----------------------- CUDA ----------------------------

	/*
    switch (mOscillatorMode) {
        case OSCILLATOR_MODE_SINE:
            for (int i = 0; i < nFrames; i++) {
                buffer[i] = sin(phaseIncArray[i]);
            }
            break;
        case OSCILLATOR_MODE_SAW:
            for (int i = 0; i < nFrames; i++) {
				buffer[i] = 1.0 - (2.0 * phaseIncArray[i] / twoPI);
            }
            break;
        case OSCILLATOR_MODE_SQUARE:
            for (int i = 0; i < nFrames; i++) {
				if (phaseIncArray[i] <= mPI) {
                    buffer[i] = 1.0;
                } else {
                    buffer[i] = -1.0;
                }
            }
            break;
        case OSCILLATOR_MODE_TRIANGLE:
            for (int i = 0; i < nFrames; i++) {
				double value = -1.0 + (2.0 * phaseIncArray[i] / twoPI);
                buffer[i] = 2.0 * (fabs(value) - 0.5);
            }
            break;
    }
	*/
	

}

/*
double Oscillator::nextSample() {
	double value = 0.0;
	if (isMuted) return value;

	switch (mOscillatorMode) {
	case OSCILLATOR_MODE_SINE:
		value = sin(mPhase);
		break;
	case OSCILLATOR_MODE_SAW:
		value = 1.0 - (2.0 * mPhase / twoPI);
		break;
	case OSCILLATOR_MODE_SQUARE:
		if (mPhase <= mPI) {
			value = 1.0;
		}
		else {
			value = -1.0;
		}
		break;
	case OSCILLATOR_MODE_TRIANGLE:
		value = -1.0 + (2.0 * mPhase / twoPI);
		value = 2.0 * (fabs(value) - 0.5);
		break;
	}
	mPhase += mPhaseIncrement;
	while (mPhase >= twoPI) {
		mPhase -= twoPI;
	}

	return value;
}
*/

double Oscillator::nextSample(double mPhase_processed) {
	double value = 0.0;
	if (isMuted) return value;

	switch (mOscillatorMode) {
	case OSCILLATOR_MODE_SINE:
		value = sin(mPhase_processed);
		break;
	case OSCILLATOR_MODE_SAW:
		value = 1.0 - (2.0 * mPhase_processed / twoPI);
		break;
	case OSCILLATOR_MODE_SQUARE:
		if (mPhase_processed <= mPI) {
			value = 1.0;
		}
		else {
			value = -1.0;
		}
		break;
	case OSCILLATOR_MODE_TRIANGLE:
		value = -1.0 + (2.0 * mPhase_processed / twoPI);
		value = 2.0 * (fabs(value) - 0.5);
		break;
	}

	/*
	mPhase += mPhaseIncrement;
	while (mPhase >= twoPI) {
		mPhase -= twoPI;
	}
	*/

	return value;
}